
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <chrono>

using namespace std;

__global__ void kernel(double *res, double *arr1, double *arr2, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;			// Абсолютный номер потока
    int offset = blockDim.x * gridDim.x;						// Общее кол-во потоков
    for(int i = idx; i < n; i += offset) {
        res[i] = arr1[i] * arr2[i];
    }
}

int main() {
    int n;
    scanf("%d", &n);
    double *res = (double *)malloc(sizeof(double) * n);
    double *vec1 = (double *)malloc(sizeof(double) * n);
    double *vec2 = (double *)malloc(sizeof(double) * n);
    for(int i = 0; i < n; i++)
        scanf("%lf", &vec1[i]);
    for(int i = 0; i < n; i++)
        scanf("%lf", &vec2[i]);

    double *dev_res, *dev_vec1, *dev_vec2;
    hipMalloc(&dev_res, sizeof(double) * n);
    hipMemcpy(dev_res, res, sizeof(double) * n, hipMemcpyHostToDevice);

    hipMalloc(&dev_vec1, sizeof(double) * n);
    hipMemcpy(dev_vec1, vec1, sizeof(double) * n, hipMemcpyHostToDevice);

    hipMalloc(&dev_vec2, sizeof(double) * n);
    hipMemcpy(dev_vec2, vec2, sizeof(double) * n, hipMemcpyHostToDevice);


    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    kernel<<<256, 256>>>(dev_res, dev_vec1, dev_vec2, n);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    fprintf(stderr, "time = %f\n", time);
    hipEventDestroy(stop);
    hipEventDestroy(start);


    hipMemcpy(res, dev_res, sizeof(double) * n, hipMemcpyDeviceToHost);
    hipFree(dev_res);
    hipFree(dev_vec1);
    hipFree(dev_vec2);
    for(int i = 0; i < n; i++) {
        printf("%f ", res[i]);
    }
    printf("\n");
    free(res);
    free(vec1);
    free(vec2);
    return 0;
}
